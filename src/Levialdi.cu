#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

#include "Levialdi.h"

__global__ void levialdiShrinkingOperator(bool* input, bool* output, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = height * col + row;

    if (input[index]) { // Case 1
        output[index] = input[index - 1] || input[index - height] || input[index - height - 1];
    }
    else { // Case 2
        output[index] = input[index - 1] && input[index - height];
    }
}

__global__ void getComponentCorners(bool* input, bool* output, int height, bool* notDone, int* corners, int* cornersNum) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = height * col + row;

    if (output[index]) {
        *notDone = true;
    } else if (input[index] && !output[index + 1] && !output[index + height] && !output[index + height + 1]) {
        // This needs to be mutex protected!
        corners[*cornersNum * 2] = col;
        corners[*cornersNum * 2 + 1] = row;
        atomicAdd(cornersNum, 1);
        // This could overflow the memory! To bad it can't be dynamic...
    }
}

std::vector<Corner> levialdiShrink(const BinaryArray& input) {

    int height = input.rows();
    int size = input.rows() * input.cols();
    bool* buffers[2];

    hipError_t err = hipSuccess;
    err = hipMalloc((void**) &buffers[0], size);
    if (err == hipSuccess) {
        err = hipMalloc((void**)&buffers[1], size);
    }
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device buffers (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(buffers[0], input.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy to device buffers (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
       
    dim3 threadsPerBlock(10, 10);
    dim3 numBlocks(input.cols() / threadsPerBlock.x, input.rows() / threadsPerBlock.y);
    int step = 0;

    std::vector<Corner> corners;
    bool* notDone = nullptr;
    int* cornersBuffer = nullptr;
    int* cornersNum = nullptr;
    hipMallocManaged(&notDone, sizeof(bool));
    hipMallocManaged(&cornersBuffer, sizeof(int) * 10);
    hipMallocManaged(&cornersNum, sizeof(int));
    *notDone = true;

    while (*notDone) {
        bool* input = buffers[step % 2];
        bool* output = buffers[(step + 1) % 2];
        levialdiShrinkingOperator<<<numBlocks, threadsPerBlock>>>(input, output, height);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch levialdiShrinkingOperator kernel (error code %s)!\n",
                hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        *notDone = false;
        *cornersNum = 0;
        getComponentCorners<<<numBlocks, threadsPerBlock>>>(input, output, height, notDone, cornersBuffer, cornersNum);
        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch getComponentCorners kernel (error code %s)!\n",
                hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
        for (int i = 0; i < *cornersNum; i++) {
            corners.push_back(Corner(cv::Point(cornersBuffer[i * 2], cornersBuffer[i * 2 + 1]), step));
        }

        step++;
    }


    hipFree(buffers[0]);
    hipFree(buffers[1]);
    hipFree(notDone);
    hipFree(cornersBuffer);
    hipFree(cornersNum);

    return corners;
}